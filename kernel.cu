
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ float A_T[TILE_SIZE][TILE_SIZE];
    __shared__ float B_T[TILE_SIZE][TILE_SIZE];



    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int p,i;
    float Pvalue = 0.0;
    for (p=0; p< (k-1)/TILE_SIZE + 1; p++) {
        
        if (row < m && p*TILE_SIZE + tx < k) {
            A_T[ty][tx] = A[row*k + p*TILE_SIZE+tx];
        }
        else {
            A_T[ty][tx] = 0.0;
        }
        if (col < n && p*TILE_SIZE+ty < k) {
            B_T[ty][tx] = B[(p*TILE_SIZE+ty)*n + col];
        }
        else {
            B_T[ty][tx] = 0.0;
        }
        __syncthreads();
        
        if (row < m && col < n)
        {
            for (i = 0; i < TILE_SIZE; i++)
                Pvalue += A_T[ty][i] * B_T[i][tx];
        }
        __syncthreads();
    }
    if (row < m && col < n){
        C[row*n + col] = Pvalue;
    }    
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid((n-1)/BLOCK_SIZE + 1,(m-1)/BLOCK_SIZE + 1,1); 
    dim3 DimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
    mysgemm<<<DimGrid,DimBlock>>>(m,n,k,A,B,C);  	
    /*************************************************************************/
}


